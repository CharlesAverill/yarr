#include "hip/hip_runtime.h"
/**
 * @file
 * @author Charles Averill
 * @date   05-Feb-2022
 * @brief Description
*/

#include "canvas.cuh"

int Canvas::save_to_ppm(char *fn)
{
    // Open file
    FILE *fp;
    fp = fopen(fn, "w+");

    if (fp == NULL) {
        return 1;
    }

    // Store canvas size
    int size = this->size;

    // Write header
    fprintf(fp, "P3 %d %d 255 ", this->width, this->height);

    for (int i = 0; i < size; i++) {
        // Skip the Alpha channel
        if (i > 0 && (i + 1) % 4 == 0) {
            continue;
        }

        fprintf(fp, "%d ", (int)this->canvas[i]);
    }

    fclose(fp);

    return 0;
}

__hd__ void Canvas::hex_int_to_color_vec(Vector<int> *out, int in)
{
    long mask1 = 255;
    long mask2 = 65280;
    long mask3 = 16711680;

    out->init((in & mask3) >> 16, (in & mask2) >> 8, in & mask1);
}

__device__ void get_sky_color(Vector<int> *color, Vector<float> ray, Canvas *canvas)
{
    canvas->hex_int_to_color_vec(color, 0xB399FF);
    (*color) = (*color) * pow(1 - ray.y, 2);
}

__device__ void
get_ground_color(Vector<int> *color, Vector<float> *ray_origin, Vector<float> ray, Canvas *canvas)
{
    float distance = -1 * ray_origin->y / ray.y;
    float x        = ray_origin->x + distance * ray.x;
    float z        = ray_origin->z + distance * ray.z;

    if ((int)abs(floor(x)) % 2 == (int)abs(floor(z)) % 2) {
        canvas->hex_int_to_color_vec(color, 0xFF0000);
    } else {
        canvas->hex_int_to_color_vec(color, 0xFFFFFF);
    }
}

__global__ void render_kernel(Canvas *canvas)
{
    // Kernel row and column based on their thread and block indices
    int x           = (threadIdx.x + blockIdx.x * blockDim.x) - (canvas->width / 2);
    int y           = (threadIdx.y + blockIdx.y * blockDim.y) - (canvas->height / 2);
    int color_index = threadIdx.z + blockIdx.z * blockDim.z;
    // The 1D index of `canvas` given our 3D information
    int index = ((y + (canvas->width / 2)) * canvas->height * canvas->channels) +
                ((x + (canvas->height / 2)) * canvas->channels) + color_index;

    // Bounds checking
    if (x >= canvas->width || y >= canvas->height || index >= canvas->size) {
        return;
    }

    // Create color vector
    Vector<int> out_color;

    // Initialize the ray
    Vector<float> ray_direction = (*(canvas->get_X()) * float(x)) +
                                  (*(canvas->get_Y()) * float(y) * -1) + (*(canvas->get_Z()));
    ray_direction            = !ray_direction;
    Vector<float> ray_origin = *(canvas->viewport_origin);

    // Cast the ray
    float hit_distance;
    Vector<float> ray_collide_position;
    Vector<float> ray_reflect_direction;
    float hit_reflectiveness;
    float ray_energy = 1.f;

    for (int reflectionIndex = 0; reflectionIndex <= MAX_REFLECTIONS; reflectionIndex++) {
        Vector<int> bounce_color;

        // Check for intersection with each triangle
        bool hit_object        = false;
        float min_hit_distance = C_INFINITY;

        RenderObjectType hit_type = RENDEROBJECT_ROT;
        Triangle *closest_triangle;
        Sphere *closest_sphere;

        for (int triangle_index = 0; triangle_index < canvas->num_triangles; triangle_index++) {
            Triangle *test_hit = &(canvas->scene_triangles)[triangle_index];
            if (is_visible(test_hit,
                           ray_origin,
                           ray_direction,
                           ray_collide_position,
                           ray_reflect_direction,
                           hit_distance,
                           bounce_color,
                           hit_reflectiveness)) {
                hit_object = true;
                if (hit_distance < min_hit_distance) {
                    min_hit_distance = hit_distance;
                    closest_triangle = test_hit;
                    hit_type         = TRIANGLE_ROT;
                }
            }
        }

        for (int sphere_index = 0; sphere_index < canvas->num_spheres; sphere_index++) {
            Sphere *test_hit = &(canvas->scene_spheres)[sphere_index];
            if (is_visible(test_hit,
                           ray_origin,
                           ray_direction,
                           ray_collide_position,
                           ray_reflect_direction,
                           hit_distance,
                           bounce_color,
                           hit_reflectiveness)) {
                hit_object = true;
                if (hit_distance < min_hit_distance) {
                    min_hit_distance = hit_distance;
                    closest_sphere   = test_hit;
                    hit_type         = SPHERE_ROT;
                }
            }
        }

        // Check for sky or ground plane
        if (hit_object) {
            switch (hit_type) {
            case TRIANGLE_ROT:
                is_visible(closest_triangle,
                           ray_origin,
                           ray_direction,
                           ray_collide_position,
                           ray_reflect_direction,
                           hit_distance,
                           bounce_color,
                           hit_reflectiveness);
                break;
            case SPHERE_ROT:
                is_visible(closest_sphere,
                           ray_origin,
                           ray_direction,
                           ray_collide_position,
                           ray_reflect_direction,
                           hit_distance,
                           bounce_color,
                           hit_reflectiveness);
                break;
            }

            ray_origin    = ray_collide_position;
            ray_direction = ray_reflect_direction;
        } else {
            if (ray_direction.y < 0) {
                get_ground_color(&bounce_color, &ray_origin, ray_direction, canvas);
                hit_reflectiveness = 0.f;
            } else {
                get_sky_color(&bounce_color, ray_direction, canvas);
                hit_reflectiveness = 0.f;
            }
        }

        // Update color and ray energy
        out_color = out_color + (bounce_color * (ray_energy * (1 - hit_reflectiveness)));
        ray_energy *= hit_reflectiveness;

        if (ray_energy <= 0.f) {
            break;
        }
    }

    // Save color data
    canvas->canvas[index]     = out_color.x;
    canvas->canvas[index + 1] = out_color.y;
    canvas->canvas[index + 2] = out_color.z;
    // Alpha
    canvas->canvas[index + 3] = 255;
}

void Canvas::scene_setup()
{
    // Initialize triangles
    thrust::host_vector<Triangle> host_triangles;

    // Octahedron
    Triangle *tri1;
    Triangle *tri2;
    Triangle *tri3;
    Triangle *tri4;
    Triangle *tri5;
    Triangle *tri6;
    Triangle *tri7;
    Triangle *tri8;

    hipMallocManaged(&tri1, sizeof(Sphere));
    hipMallocManaged(&tri2, sizeof(Sphere));
    hipMallocManaged(&tri3, sizeof(Sphere));
    hipMallocManaged(&tri4, sizeof(Sphere));
    hipMallocManaged(&tri5, sizeof(Sphere));
    hipMallocManaged(&tri6, sizeof(Sphere));
    hipMallocManaged(&tri7, sizeof(Sphere));
    hipMallocManaged(&tri8, sizeof(Sphere));

    init_triangle(tri1,
                  Vector<float>(0, 0, 0),
                  Vector<float>(-1, 1, 0),
                  Vector<float>(0, 1, 1),
                  Vector<int>(0, 0, 0),
                  0.95f);

    init_triangle(tri2,
                  Vector<float>(0, 0, 0),
                  Vector<float>(0, 1, -1),
                  Vector<float>(-1, 1, 0),
                  Vector<int>(0, 0, 0),
                  0.95f);

    init_triangle(tri3,
                  Vector<float>(0, 0, 0),
                  Vector<float>(1, 1, 0),
                  Vector<float>(0, 1, -1),
                  Vector<int>(0, 0, 0),
                  0.95f);

    init_triangle(tri4,
                  Vector<float>(0, 0, 0),
                  Vector<float>(0, 1, 1),
                  Vector<float>(1, 2, 0),
                  Vector<int>(0, 0, 0),
                  0.95f);

    init_triangle(tri5,
                  Vector<float>(0, 2, 0),
                  Vector<float>(0, 1, 1),
                  Vector<float>(-1, 1, 0),
                  Vector<int>(0, 0, 0),
                  0.95f);

    init_triangle(tri6,
                  Vector<float>(0, 2, 0),
                  Vector<float>(1, 1, 0),
                  Vector<float>(0, 1, 1),
                  Vector<int>(0, 0, 0),
                  0.95f);

    init_triangle(tri7,
                  Vector<float>(0, 2, 0),
                  Vector<float>(0, 1, -1),
                  Vector<float>(1, 1, 0),
                  Vector<int>(0, 0, 0),
                  0.95f);

    init_triangle(tri8,
                  Vector<float>(0, 2, 0),
                  Vector<float>(-1, 1, 0),
                  Vector<float>(0, 1, -1),
                  Vector<int>(0, 0, 0),
                  0.95f);

    host_triangles.push_back(*tri1);
    host_triangles.push_back(*tri2);
    host_triangles.push_back(*tri3);
    host_triangles.push_back(*tri4);
    host_triangles.push_back(*tri5);
    host_triangles.push_back(*tri6);
    host_triangles.push_back(*tri7);
    host_triangles.push_back(*tri8);

    // Initialize Spheres
    thrust::host_vector<Sphere> host_spheres;

    Sphere *reflective_sphere;
    hipMallocManaged(&reflective_sphere, sizeof(Sphere));
    init_sphere(reflective_sphere, Vector<float>(1, 2, 0), 0.5f, Vector<int>(0, 0, 0), 0.95f);
    host_spheres.push_back(*reflective_sphere);

    Sphere *red_sphere;
    hipMallocManaged(&red_sphere, sizeof(Sphere));
    init_sphere(red_sphere, Vector<float>(-1.25f, 0.8f, 0), 0.25f, Vector<int>(255, 0, 0), 0.5f);
    host_spheres.push_back(*red_sphere);

    // Copy triangles to device
    hipMallocManaged(&(this->scene_triangles), sizeof(Triangle) * host_triangles.size());
    hipMemcpy(this->scene_triangles,
               thrust::raw_pointer_cast(host_triangles.data()),
               sizeof(Triangle) * host_triangles.size(),
               hipMemcpyHostToDevice);
    this->num_triangles = host_triangles.size();

    // Copy spheres to device
    hipMallocManaged(&(this->scene_spheres), sizeof(Sphere) * host_spheres.size());
    hipMemcpy(this->scene_spheres,
               thrust::raw_pointer_cast(host_spheres.data()),
               sizeof(Sphere) * host_spheres.size(),
               hipMemcpyHostToDevice);
    this->num_spheres = host_spheres.size();
}

void Canvas::render()
{
    // Run render kernel
    render_kernel<<<this->grid_size, this->block_size>>>(this);
}
