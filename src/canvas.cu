#include "hip/hip_runtime.h"
/**
 * @file
 * @author Charles Averill
 * @date   05-Feb-2022
 * @brief Description
*/

#include "canvas.cuh"

int Canvas::save_to_ppm(char *fn)
{
    // Open file
    FILE *fp;
    fp = fopen(fn, "w+");

    if (fp == NULL) {
        return 1;
    }

    // Store canvas size
    int size = this->size;

    // Write header
    fprintf(fp, "P3 %d %d 255 ", this->width, this->height);

    for (int i = 0; i < size; i++) {
        // Skip the Alpha channel
        if (i > 0 && (i + 1) % 4 == 0) {
            continue;
        }

        fprintf(fp, "%d ", (int)this->canvas[i]);
    }

    fclose(fp);

    return 0;
}

__hd__ void Canvas::hex_int_to_color_vec(Vector<int> *out, int in)
{
    long mask1 = 255;
    long mask2 = 65280;
    long mask3 = 16711680;

    out->init((in & mask3) >> 16, (in & mask2) >> 8, in & mask1);
}

__device__ void get_sky_color(Vector<int> *color, Vector<float> ray, Canvas *canvas)
{
    canvas->hex_int_to_color_vec(color, 0xB399FF);
    (*color) = (*color) * pow(1 - ray.y, 2);
}

__device__ void
get_ground_color(Vector<int> *color, Vector<float> *ray_origin, Vector<float> ray, Canvas *canvas)
{
    float distance = -1 * ray_origin->y / ray.y;
    float x = ray_origin->x + distance * ray.x;
    float z = ray_origin->z + distance * ray.z;

    if ((int)abs(floor(x)) % 2 == (int)abs(floor(z)) % 2) {
        canvas->hex_int_to_color_vec(color, 0xFF0000);
    } else {
        canvas->hex_int_to_color_vec(color, 0xFFFFFF);
    }
}

__global__ void render_kernel(Canvas *canvas)
{
    // Kernel row and column based on their thread and block indices
    int x = (threadIdx.x + blockIdx.x * blockDim.x) - (canvas->width / 2);
    int y = (threadIdx.y + blockIdx.y * blockDim.y) - (canvas->height / 2);
    int color_index = threadIdx.z + blockIdx.z * blockDim.z;
    // The 1D index of `canvas` given our 3D information
    int index = ((y + (canvas->width / 2)) * canvas->height * canvas->channels) +
                ((x + (canvas->height / 2)) * canvas->channels) + color_index;

    // Bounds checking
    if (x >= canvas->width || y >= canvas->height || index >= canvas->size) {
        return;
    }

    // Create color vector
    Vector<int> out_color;

    // Initialize the ray
    Vector<float> ray_direction = (*(canvas->get_X()) * float(x)) +
                                  (*(canvas->get_Y()) * float(y) * -1) + (*(canvas->get_Z()));
    ray_direction = !ray_direction;
    Vector<float> ray_origin = *(canvas->viewport_origin);

    // Cast the ray
    float hit_distance;
    Vector<float> ray_collide_position;
    Vector<float> ray_reflect_direction;
    float hit_reflectiveness;
    float ray_energy = 1.f;

    for (int reflectionIndex = 0; reflectionIndex <= MAX_REFLECTIONS; reflectionIndex++) {
        Vector<int> bounce_color;

        // Check for intersection with each triangle
        bool hit_object = false;
        float min_hit_distance = C_INFINITY;

        RenderObject *closest_renderobject;

        for (int index = 0; index < canvas->num_renderobjects; index++) {
            RenderObject *test_hit = canvas->scene_renderobjects[index];
            if (test_hit->is_visible(ray_origin,
                                     ray_direction,
                                     ray_collide_position,
                                     ray_reflect_direction,
                                     hit_distance,
                                     bounce_color,
                                     hit_reflectiveness)) {
                hit_object = true;
                if (hit_distance < min_hit_distance) {
                    min_hit_distance = hit_distance;
                    closest_renderobject = test_hit;
                }
            }
        }

        // Check for sky or ground plane
        if (hit_object && closest_renderobject) {
            closest_renderobject->is_visible(ray_origin,
                                             ray_direction,
                                             ray_collide_position,
                                             ray_reflect_direction,
                                             hit_distance,
                                             bounce_color,
                                             hit_reflectiveness);

            ray_origin = ray_collide_position;
            ray_direction = ray_reflect_direction;
        } else {
            if (ray_direction.y < 0) {
                get_ground_color(&bounce_color, &ray_origin, ray_direction, canvas);
                hit_reflectiveness = 0.f;
            } else {
                get_sky_color(&bounce_color, ray_direction, canvas);
                hit_reflectiveness = 0.f;
            }
        }

        // Update color and ray energy
        out_color = out_color + (bounce_color * (ray_energy * (1 - hit_reflectiveness)));
        ray_energy *= hit_reflectiveness;

        if (ray_energy <= 0.f) {
            break;
        }
    }

    // Save color data
    canvas->canvas[index] = out_color.x;
    canvas->canvas[index + 1] = out_color.y;
    canvas->canvas[index + 2] = out_color.z;
    // Alpha
    canvas->canvas[index + 3] = 255;
}

__global__ void scene_setup_kernel(Canvas *canvas)
{
    // Initialize triangles
    List<RenderObject *> renderobjects;

    // Octahedron
    Octahedron *oct = new Octahedron(Vector<float>{0, 1, 0}, 1.0f);
    oct->extend_list(&renderobjects);

    // Initialize Spheres
    renderobjects.add(new Sphere(Vector<float>{1, 2, 0}, 0.5f, Vector<int>{0, 0, 0}, 0.95f));
    renderobjects.add(
        new Sphere(Vector<float>{-1.25, 0.8, 0}, 0.25f, Vector<int>{255, 0, 0}, 0.5f));

    // Copy triangles to device
    canvas->scene_renderobjects =
        (RenderObject **)malloc(sizeof(RenderObject *) * renderobjects.size());
    //hipMallocManaged(&(canvas->scene_triangles), sizeof(Triangle) * host_triangles.size());
    hipMemcpyAsync(canvas->scene_renderobjects,
                    renderobjects.getArray(),
                    sizeof(RenderObject *) * renderobjects.size(),
                    hipMemcpyDeviceToDevice);
    canvas->num_renderobjects = renderobjects.size();
}

void Canvas::scene_setup()
{
    scene_setup_kernel<<<1, 1>>>(this);
    gpuErrorCheck(hipPeekAtLastError());
    gpuErrorCheck(hipDeviceSynchronize());
}

void Canvas::render()
{
    // Run render kernel
    render_kernel<<<this->grid_size, this->block_size>>>(this);
    gpuErrorCheck(hipPeekAtLastError());
    gpuErrorCheck(hipDeviceSynchronize());
}
