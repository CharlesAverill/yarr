#include "hip/hip_runtime.h"
/**
 * @file
 * @author Charles Averill
 * @date   04-Feb-2022
 * @brief Description
*/

#include <SFML/Graphics.h>
#include <math.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>

#include "canvas.cuh"
#include "input.cuh"
#include "utils/cuda_utils.cuh"
#include "utils/utils.cuh"
#include "utils/vector.cuh"

#define BLOCK_SIZE 16

sfRenderWindow *csfml_setup(unsigned int width, unsigned int height)
{
    sfVideoMode mode = {width, height, 32};

    sfRenderWindow *window = sfRenderWindow_create(mode, "YARR", sfResize | sfClose, NULL);
    if (!window) {
        fprintf(stderr, "[csfml_setup] Couldn't initialize SFML window");
        exit(1);
    }

    sfRenderWindow_setFramerateLimit(window, 60);

    return window;
}

void render_loop(Canvas *canvas, sfRenderWindow *window)
{
    sfEvent event;
    sfTexture *texture;
    sfSprite *sprite;

    texture = sfTexture_create(canvas->width, canvas->height);
    if (!texture) {
        return;
    }

    sprite = sfSprite_create();
    sfSprite_setTexture(sprite, texture, sfTrue);

    while (sfRenderWindow_isOpen(window)) {
        // Process Input
        input_loop(window, &event);

        // Update Scene
        //*(canvas->viewport_origin) = *(canvas->viewport_origin) + Vector<float>(0.1f, 0, 0);

        // Render output
        canvas->render();

        sfRenderWindow_clear(window, sfBlack);

        sfTexture_updateFromPixels(texture, canvas->canvas, canvas->width, canvas->height, 0, 0);
        sfRenderWindow_drawSprite(window, sprite, NULL);

        sfRenderWindow_display(window);
    }

    sfSprite_destroy(sprite);
    sfTexture_destroy(texture);
    sfRenderWindow_destroy(window);
}

int main(int argc, char *argv[])
{
    // Deal with input arguments
    const char *output_fn;
    if (argc < 2) {
        output_fn = "yarr.ppm";
    } else {
        output_fn = argv[1];
    }

    // These are the dimensions of a 4D (RGBA) matrix that we will flatten into 1D
    int width = 512;
    int height = 512;
    int channels = 4;

    // Get device information from CUDA
    int device_ID;
    hipDeviceProp_t props;

    hipGetDevice(&device_ID);
    hipGetDeviceProperties(&props, device_ID);

    // Calculate our kernel dimensions
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 grid_size(int(ceil(float(width) / float(BLOCK_SIZE))),
                   int(ceil(float(height) / float(BLOCK_SIZE))),
                   1);

    // Instantiate our Canvas object
    Canvas *canvas;
    hipMallocManaged(&canvas, sizeof(Canvas));
    canvas->init(width, height, channels);

    // Setup scene
    canvas->host_setup(grid_size, block_size);

    // Setup window
    sfRenderWindow *window = csfml_setup(width, height);

    // Call render loop here
    render_loop(canvas, window);

    // Save last render to PPM
    fprintf(stdout, "[main] Saving last render to %s\n", output_fn);
    canvas->save_to_ppm(output_fn);

    // Free memory
    hipFree(canvas);
}
