#include "hip/hip_runtime.h"
/**
 * @file
 * @author Charles Averill
 * @date   04-Feb-2022
 * @brief Description
*/

#include <math.h>
#include <stdio.h>

#include "canvas.cuh"
#include "cuda_utils.cuh"
#include "utils.cuh"
#include "vector.cuh"

#define BLOCK_SIZE 16

int main(int argc, char *argv[])
{
    // Deal with input arguments
    char *output_fn;
    if (argc < 2) {
        output_fn = "yarr.ppm";
    } else {
        output_fn = argv[1];
    }

    // These are the dimensions of a 3D matrix that we will flatten into 1D
    int width    = 512;
    int height   = 512;
    int channels = 3;

    // Get device information from CUDA
    int device_ID;
    hipDeviceProp_t props;

    hipGetDevice(&device_ID);
    hipGetDeviceProperties(&props, device_ID);

    // Calculate our kernel dimensions
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 grid_size(int(ceil(float(width) / float(BLOCK_SIZE))),
                   int(ceil(float(height) / float(BLOCK_SIZE))),
                   1);

    // Instantiate our Canvas object
    Canvas *canvas;
    hipMallocManaged(&canvas, sizeof(Canvas));
    canvas->init(width, height, channels);

    // Initialize our canvas on the GPU
    canvas->render(grid_size, block_size);

    // Synchronize and check for errors
    gpuErrorCheck(hipPeekAtLastError());
    gpuErrorCheck(hipDeviceSynchronize());

    // Save canvas to PPM
    printf("Saving render to %s\n", output_fn);
    canvas->save_to_ppm(output_fn);

    // Free memory
    hipFree(canvas);
}
