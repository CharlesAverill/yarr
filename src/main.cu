#include "hip/hip_runtime.h"
/**
 * @file
 * @author Charles Averill
 * @date   04-Feb-2022
 * @brief Description
*/

#include <stdio.h>
#include <math.h>

#include "utils.h"
#include "canvas.h"

#define BLOCK_SIZE 16

__global__ void init_canvas(canvas *c, int c_size) {
    // Kernel row and column based on their thread and block indices
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    // The 1D index of `canvas` given our 3D information
    // This represents a pixel, so we will still loop 3 times to fill out RGB information
    int index = (row * c->height * c->channels) + (col * c->channels) + z;

    if (row >= c->width || col >= c->height || index >= c_size) {
        return;
    }

    c->values[index] = (z == 2) ? 255 : 0;
}

int main(int argc, char *argv[]) {
    // Deal with input arguments
    char *output_fn;
    if(argc < 2) {
        output_fn = "yarr.ppm";
    } else {
        output_fn = argv[1];
    }

    // These are the dimensions of a 3D matrix that we will flatten into 1D
    int width = 512;
    int height = 512;
    int channels = 3;

    // Initialize our canvas struct
    canvas *c;
    hipMallocManaged(&c, sizeof(canvas));
    // I'm not yet sure why I need to multiply the size by 4 here, but without it I run into
    // GPUassert: an illegal memory access was encountered
    hipMallocManaged(&(c->values), width * height * channels * 4);
    c->width = width;
    c->height = height;
    c->channels = channels;

    // Get device information from CUDA
    int device_ID;
    hipDeviceProp_t props;

    hipGetDevice(&device_ID);
    hipGetDeviceProperties(&props, device_ID);

    // Calculate our kernel dimensions
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 grid_size(int(ceil(float(width) / float(BLOCK_SIZE))), int(ceil(float(height) / float(BLOCK_SIZE))), channels);

    printf("%d %d %d\n%d %d %d\n", block_size.x, block_size.y, block_size.z, grid_size.x, grid_size.y, grid_size.z);

    // Initialize our canvas on the GPU
    init_canvas<<<grid_size, block_size>>>(c, canvas_size(c));

    // Synchronize and check for errors
    gpuErrorCheck(hipPeekAtLastError());
    gpuErrorCheck(hipDeviceSynchronize());

    // Save canvas to PPM
    canvas_to_ppm(c, output_fn);

    // Free memory
    hipFree(c);
}
